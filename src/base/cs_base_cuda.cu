#include "hip/hip_runtime.h"
/*============================================================================
 * Low-level functions and global variables definition for CUDA.
 *============================================================================*/

/*
  This file is part of Code_Saturne, a general-purpose CFD tool.

  Copyright (C) 1998-2020 EDF S.A.

  This program is free software; you can redistribute it and/or modify it under
  the terms of the GNU General Public License as published by the Free Software
  Foundation; either version 2 of the License, or (at your option) any later
  version.

  This program is distributed in the hope that it will be useful, but WITHOUT
  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
  FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more
  details.

  You should have received a copy of the GNU General Public License along with
  this program; if not, write to the Free Software Foundation, Inc., 51 Franklin
  Street, Fifth Floor, Boston, MA 02110-1301, USA.
*/

/*----------------------------------------------------------------------------*/

#include "cs_defs.h"

/*----------------------------------------------------------------------------
 * Standard C library headers
 *----------------------------------------------------------------------------*/

/*----------------------------------------------------------------------------
 * Local headers
 *----------------------------------------------------------------------------*/

#include "bft_error.h"
#include "bft_mem.h"
#include "bft_printf.h"

#include "cs_base.h"
#include "cs_log.h"

/*----------------------------------------------------------------------------
 *  Header for the current file
 *----------------------------------------------------------------------------*/

#include "cs_base_cuda.h"

/*----------------------------------------------------------------------------*/

BEGIN_C_DECLS

/*! \cond DOXYGEN_SHOULD_SKIP_THIS */

/*============================================================================
 * Local Macro Definitions
 *============================================================================*/

#define CS_CUDA_CHECK(x)                                                       \
if (hipError_t err = (x)) {                                                   \
  bft_error(__FILE__, __LINE__, 0, _("CUDA error: %s"), hipGetErrorString(err)); \
}

/*============================================================================
 * Local Type Definitions
 *============================================================================*/

/*============================================================================
 *  Global variables
 *============================================================================*/

int  cs_glob_cuda_device_id = -1;

/*============================================================================
 * Private function definitions
 *============================================================================*/

/*! (DOXYGEN_SHOULD_SKIP_THIS) \endcond */

/*============================================================================
 * Public function definitions
 *============================================================================*/

/*----------------------------------------------------------------------------*/
/*!
 * \brief  Log information on available CUDA devices.
 *
 * \param[in]  log_id  id of log file in which to print information
 */
/*----------------------------------------------------------------------------*/

void
cs_base_cuda_device_info(cs_log_t  log_id)
{
  int n_devices = 0;

  hipError_t retval = hipGetDeviceCount(&n_devices);

  if (retval == hipErrorNoDevice)
    cs_log_printf(log_id,
                  _("  CUDA device:         none available\n"));
  else if (retval)
    cs_log_printf(log_id,
                  _("  CUDA device:         %s\n"),
		  hipGetErrorString(retval));

  char buffer[256] = "";

  for (int i = 0; i < n_devices; i++) {
    struct hipDeviceProp_t prop;
    CS_CUDA_CHECK(hipGetDeviceProperties(&prop, i));
    unsigned long long mem = prop.totalGlobalMem / 1000000;
    char mode_name[32] = "";
    if (prop.computeMode == hipComputeModeDefault)
      snprintf(mode_name, 31, "default");
    else if (prop.computeMode == hipComputeModeExclusive)
      snprintf(mode_name, 31, "exclusive");
    else if (prop.computeMode == hipComputeModeProhibited)
      snprintf(mode_name, 31, "prohibited");

    cs_log_printf
      (log_id,
       _("  CUDA device %d:       %s\n"),
       i, prop.name);

    if (strncmp(prop.name, buffer, 255) != 0)
      cs_log_printf
        (log_id,
         _("                       Compute capability: %d.%d\n"
           "                       Memory: %llu %s\n"
           "                       Integrated: %d\n"
           "                       Can map host memory: %d\n"
           "                       Compute mode: %s\n"),
         prop.major, prop.minor,
         mem, _("MB"),
         prop.integrated,
         prop.canMapHostMemory, mode_name);

    strncpy(buffer, prop.name, 255);
    buffer[255] = '\0';
  }
}

/*----------------------------------------------------------------------------*/

END_C_DECLS
